#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

__global__ void mem_transfer_test(int* input)
{
	int grid_id = blockIdx.x * blockDim.x + threadIdx.x;
	printf("thread ID: %d, grid ID: %d, value: %d\n", threadIdx.x, grid_id, input[grid_id]);
}

__global__ void mem_transfer_test2(int* input, int size)
{
	int grid_id = blockIdx.x * blockDim.x + threadIdx.x;

	if (grid_id < size)
		printf("thread ID: %d, grid ID: %d, value: %d\n", threadIdx.x, grid_id, input[grid_id]);
}

int main()
{
	int size = 150;
	int byte_size = size * sizeof(int);

	int* host_input;
	host_input = (int*)malloc(byte_size);

	time_t t;
	srand((unsigned)time(&t));

	for (int i = 0; i < size; i++)
	{
		host_input[i] = (int)(rand() & 0xff);
	}

	int* device_input;
	hipMalloc((void**)&device_input, byte_size);
	hipMemcpy(device_input, host_input, byte_size, hipMemcpyHostToDevice);

	dim3 block(32);
	dim3 grid(5);

	mem_transfer_test2<<<grid , block>>>(device_input, size);
	hipDeviceSynchronize();

	hipFree(device_input);
	free(host_input);

	hipDeviceReset();
	return 0;
}