#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void print_details_exercise()
{
	printf("threadIdx.x : %d, threadIdx.y : %d, threadIdx.z :%d, blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, blockDim.x : %d, blockDim.y : %d, gridDim.x : %d, gridDim.y : %d\n",
		threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
}

/*
int main()
{
	int nx = 4, ny = 4, nz = 4;

	dim3 block(2, 2, 2);
	dim3 grid(nx / block.x, ny / block.y, nz / block.z);
	
	print_details_exercise<<<grid, block>>>();
	cudaDeviceSynchronize();

	cudaDeviceReset();
	return 0;
}
*/