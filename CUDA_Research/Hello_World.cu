#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void hello_cuda()
{
	printf("Hello World from CUDA!\n");
}

int main()
{
	int nx = 16, ny = 4;

	//Limit for block size x <= 65536, y <= 2^32-1, z <= 65536
	dim3 block(8, 2); //Number of Dimension 3 blocks/threads within each grid

	//Limit for grid size x <= 1024, y <= 1024, z <= 64
	//AND
	//Limit for grid size x * y * z <= 1024
	dim3 grid(nx / block.x, ny / block.y); //Number of Dimension 3 grids on the GPU

	hello_cuda <<<grid, block>>>();
	hipDeviceSynchronize(); //Call above is async without this function call!

	hipDeviceReset();
	return 0;
}