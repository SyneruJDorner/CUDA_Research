#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void hello_cuda()
{
	printf("Hello World from CUDA!\n");
}

int main()
{
	int nx = 16, ny = 4;

	dim3 block(8, 2); //Number of Dimension 3 blocks/threads within each grid
	dim3 grid(nx / block.x, ny / block.y); //Number of Dimension 3 grids on the GPU

	hello_cuda <<<grid, block>>>();
	hipDeviceSynchronize(); //Call above is async without this function call!

	hipDeviceReset();
	return 0;
}