#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void hello_cuda()
{
	printf("Hello World from CUDA!");
}

int main()
{
	//How to call the cuda function
	//P1 = 
	//P2 = number of threads to the GPU that is called.
	hello_cuda <<<1, 1>>>();
	hipDeviceSynchronize(); //Call above is async without this function call!

	hipDeviceReset();
	return 0;
}