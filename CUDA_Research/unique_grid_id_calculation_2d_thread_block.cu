#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void unique_grid_id_calculation_2d_2d(int* data)
{
	int thread_id = blockDim.x * threadIdx.y + threadIdx.x;

	int num_threads_in_a_block = blockDim.x * blockDim.y;
	int block_offset = blockIdx.x * num_threads_in_a_block;

	int num_threads_in_a_row = num_threads_in_a_block * gridDim.x;
	int row_offset = num_threads_in_a_row * blockIdx.y;

	int grid_id = thread_id + row_offset + block_offset;

	printf("blockIdx.x: %d, blockIdx.y: %d, threadIdx.x: %d, grid ID: %d, - data : %d \n",
		blockIdx.x, blockIdx.y, thread_id, grid_id, data[grid_id]);
}

int main()
{
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 23, 9, 4, 53, 65, 12, 1, 33, 87, 45, 23, 12, 342, 56, 44, 99 };

	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(2, 2);
	dim3 grid(2, 2);

	unique_grid_id_calculation_2d_2d<<<grid, block>>>(d_data);
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}