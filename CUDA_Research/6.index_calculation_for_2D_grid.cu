#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void unique_grid_id_calculation_2d(int* data)
{
	int thread_id = threadIdx.x;
	int block_offset = blockIdx.x * blockDim.x;
	int row_offset = blockDim.x * gridDim.x * blockIdx.y;
	int grid_id = row_offset + block_offset + thread_id;
	printf("blockIdx.x: %d, blockIdx.y: %d, threadIdx.x: %d, grid ID: %d, - data : %d \n",
		blockIdx.x, blockIdx.y, thread_id, grid_id, data[grid_id]);
}

/*
int main()
{
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 23, 9, 4, 53, 65, 12, 1, 33, 87, 45, 23, 12, 342, 56, 44, 99 };

	int* d_data;
	cudaMalloc((void**)&d_data, array_byte_size);
	cudaMemcpy(d_data, h_data, array_byte_size, cudaMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(2, 2);

	unique_grid_id_calculation_2d<<<grid, block>>>(d_data);
	cudaDeviceSynchronize();

	cudaDeviceReset();
	return 0;
}
*/