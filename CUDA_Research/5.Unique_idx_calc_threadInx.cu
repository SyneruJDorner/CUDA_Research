#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void unique_idx_calc_threadInx(int* input)
{
	int thread_id = threadIdx.x;
	printf("threadIdx: %d, value: %d \n", thread_id, input[thread_id]);
}

__global__ void unique_grid_calculation(int* input)
{
	int thread_id = threadIdx.x;
	int offset = blockIdx.x * blockDim.x;
	int grid_id = thread_id + offset;
	printf("blockIdx.x: %d, threadIdx: %d, gid: %d, value: %d \n",
		blockIdx.x, thread_id, grid_id, input[grid_id]);
}

/*
int main()
{
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 23, 9, 4, 53, 65, 12, 1, 33, 87, 45, 23, 12, 342, 56, 44, 99 };

	for (int i = 0; i < array_size; i++)
	{
		printf("%d ", h_data[i]);
	}

	printf("\n\n");

	int* d_data;
	cudaMalloc((void**)&d_data, array_byte_size);
	cudaMemcpy(d_data, h_data, array_byte_size, cudaMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(4);

	unique_grid_calculation<<<grid, block>>>(d_data);
	cudaDeviceSynchronize();

	cudaDeviceReset();
	return 0;
}
*/